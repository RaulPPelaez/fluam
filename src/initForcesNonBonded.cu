#include "hip/hip_runtime.h"
// Filename: initForcesNonBonded.cu
//
// Copyright (c) 2010-2015, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


float functionForceNonBonded1(double r){
  float sigma, epsilon;
  sigma = 2 * lx / double(mx);
  epsilon = temperature ;
  //return -epsilon * ( r - sigma);
  return 48. * epsilon * (pow(sigma/r,12) - 0.5*pow(sigma/r,6))/r;


}

bool initForcesNonBonded(){
  texforceNonBonded1.normalized = true;
  texforceNonBonded1.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texforceNonBonded1.filterMode = hipFilterModeLinear;//hipFilterModeLinear and hipFilterModePoint

  float *h_data;
 
  int ntypes = 1;
  double *Aij_param;
  double *Bij_param;
  if(loadparticles==0){
    Aij_param = new double[ntypes*ntypes];
    Bij_param = new double[ntypes*ntypes];
    for(int i=0; i<ntypes; i++){
      for(int j=0; j<ntypes; j++){
        float sigma, epsilon;
        sigma = 2 * lx / float(mx); //READ FROM FILE
        epsilon = temperature ;
        Aij_param[i+ntypes*j] = 48.0f * pow(sigma,12)*epsilon;
        Bij_param[i+ntypes*j] = 48.0f * pow(sigma,6)*0.5*epsilon;
      }
    }
  }
   else{
     ifstream in("LJ.in");
     in>>ntypes;
     Aij_param = new double[ntypes*ntypes];
     Bij_param = new double[ntypes*ntypes];
     for(int i=0; i<ntypes; i++){
      for(int j=0; j<ntypes; j++){
        in>>Aij_param[i+ntypes*j];
      }
     }
     for(int i=0; i<ntypes; i++){
      for(int j=0; j<ntypes; j++){
        in>>Bij_param[i+ntypes*j];
      }
     }

  hipMalloc((void **)&Aij_paramGPU, ntypes*ntypes*sizeof(double));
  hipMalloc((void **)&Bij_paramGPU, ntypes*ntypes*sizeof(double));

  hipMemcpy(Aij_paramGPU, Aij_param, ntypes*ntypes*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(Bij_paramGPU, Bij_param, ntypes*ntypes*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(ntypesGPU), &ntypes, sizeof(int));





  int size = 4096;
  h_data = new float[size];
  float r, dr;
  float cutoff2 = cutoff * cutoff;
  dr = cutoff2/float(size);
  r = 0.5 * dr;
  for(int i=0;i<size;i++){
    h_data[i] = functionForceNonBonded1(sqrt(r))/sqrt(r);
    //cout << sqrt(r) << "   " << h_data[i] << endl;
    r += dr;
  }
  h_data[size-1] = 0.;
  h_data[0] = 0.;
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  cutilSafeCall( hipMallocArray( &forceNonBonded1, &channelDesc, size, 1 )); 
  cutilSafeCall( hipMemcpyToArray( forceNonBonded1, 0, 0, h_data, size*sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall( hipBindTextureToArray( texforceNonBonded1, forceNonBonded1, channelDesc));


  /*r = 0.5 * dr;
    for(int i=0;i<size;i++){
    cout << r << " " << h_data[i] << endl;
    r += dr;
    }*/
  cout << "INIT FORCE NON-BONDED 1 COMPLETED" << endl;
  delete[] h_data;
  delete[] Aij_param;
  delete[] Bij_param;

  return 1;
}
