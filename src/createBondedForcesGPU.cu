#include "hip/hip_runtime.h"
// Filename: createBondedForcesGPU.cu
//
// Copyright (c) 2010-2015, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


__global__ void initBondedForcesVariables(bondedForcesVariables* bFV,
					  int* bondsParticleParticleGPU,
					  int* bondsParticleParticleOffsetGPU,
					  int* bondsIndexParticleParticleGPU,
					  double* r0ParticleParticleGPU,
					  double* kSpringParticleParticleGPU,
					  int* bondsParticleFixedPointGPU,
					  int* bondsParticleFixedPointOffsetGPU,
					  double* r0ParticleFixedPointGPU,
					  double* kSpringParticleFixedPointGPU,
					  double* rxFixedPointGPU,
					  double* ryFixedPointGPU,
					  double* rzFixedPointGPU){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>0) return;   

  bFV->bondsParticleParticleGPU       = bondsParticleParticleGPU;
  bFV->bondsParticleParticleOffsetGPU = bondsParticleParticleOffsetGPU;
  bFV->bondsIndexParticleParticleGPU  = bondsIndexParticleParticleGPU;
  bFV->r0ParticleParticleGPU          = r0ParticleParticleGPU;
  bFV->kSpringParticleParticleGPU     = kSpringParticleParticleGPU;


  bFV->bondsParticleFixedPointGPU       = bondsParticleFixedPointGPU;
  bFV->bondsParticleFixedPointOffsetGPU = bondsParticleFixedPointOffsetGPU;
  bFV->r0ParticleFixedPointGPU          = r0ParticleFixedPointGPU;
  bFV->kSpringParticleFixedPointGPU     = kSpringParticleFixedPointGPU;
  bFV->rxFixedPointGPU          = rxFixedPointGPU;
  bFV->ryFixedPointGPU          = ryFixedPointGPU;
  bFV->rzFixedPointGPU          = rzFixedPointGPU;

}




bool createBondedForcesGPU(){

  //Copy constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(bondedForcesGPU),&bondedForces,sizeof(bool));

  //Allocate memory
  cutilSafeCall(hipMalloc((void**)&bFV,sizeof(bondedForcesVariables)));
  cutilSafeCall(hipMalloc((void**)&bondsParticleParticleGPU,np*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&bondsParticleParticleOffsetGPU,np*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&bondsIndexParticleParticleGPU,nbondsParticleParticle*2*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&r0ParticleParticleGPU,nbondsParticleParticle*2*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&kSpringParticleParticleGPU,nbondsParticleParticle*2*sizeof(double)));

  //Copy global memory
  cutilSafeCall(hipMemcpy(bondsParticleParticleGPU,bondsParticleParticle,
			   np*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(bondsParticleParticleOffsetGPU,bondsParticleParticleOffset,
			   np*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(bondsIndexParticleParticleGPU,bondsIndexParticleParticle,
			   nbondsParticleParticle*2*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(r0ParticleParticleGPU,r0ParticleParticle,
			   nbondsParticleParticle*2*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(kSpringParticleParticleGPU,kSpringParticleParticle,
			   nbondsParticleParticle*2*sizeof(double),hipMemcpyHostToDevice));





  //Allocate memory
  cutilSafeCall(hipMalloc((void**)&bondsParticleFixedPointGPU,np*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&bondsParticleFixedPointOffsetGPU,np*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&r0ParticleFixedPointGPU,nbondsParticleFixedPoint*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&kSpringParticleFixedPointGPU,nbondsParticleFixedPoint*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rxFixedPointGPU,nbondsParticleFixedPoint*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&ryFixedPointGPU,nbondsParticleFixedPoint*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rzFixedPointGPU,nbondsParticleFixedPoint*sizeof(double)));

  //Copy global memory
  cutilSafeCall(hipMemcpy(bondsParticleFixedPointGPU,bondsParticleFixedPoint,
			   np*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(bondsParticleFixedPointOffsetGPU,bondsParticleFixedPointOffset,
			   np*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(r0ParticleFixedPointGPU,r0ParticleFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(kSpringParticleFixedPointGPU,kSpringParticleFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(rxFixedPointGPU,rxFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(ryFixedPointGPU,ryFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(rzFixedPointGPU,rzFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice));




  initBondedForcesVariables<<<1,1>>>(bFV,
				     bondsParticleParticleGPU,
				     bondsParticleParticleOffsetGPU,
				     bondsIndexParticleParticleGPU,
				     r0ParticleParticleGPU,
				     kSpringParticleParticleGPU,
				     bondsParticleFixedPointGPU,
				     bondsParticleFixedPointOffsetGPU,
				     r0ParticleFixedPointGPU,
				     kSpringParticleFixedPointGPU,
				     rxFixedPointGPU,
				     ryFixedPointGPU,
				     rzFixedPointGPU);




  return 1;
}
