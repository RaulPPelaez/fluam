#include "hip/hip_runtime.h"
__global__ void findNeighborListsQuasi2D(particlesincell* pc, 
					 int* errorKernel,
					 const double* rxcellGPU,
					 const double* rycellGPU,
					 const double* rzcellGPU,
					 const double* rxboundaryGPU,  //q^{n}
					 const double* ryboundaryGPU, 
					 const double* rzboundaryGPU){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=(npGPU)) return;   
  
  double rx = fetch_double(texrxboundaryGPU,nboundaryGPU+i);
  double ry = fetch_double(texryboundaryGPU,nboundaryGPU+i);

  int icel, np;
 
  // Particle location in cells for neighbor lists
  {
    double invdx = double(mxNeighborsGPU)/lxGPU;
    double invdy = double(myNeighborsGPU)/lyGPU;
    rx = rx - (int(rx*invlxGPU + 0.5*((rx>0)-(rx<0)))) * lxGPU;
    int jx   = int(rx * invdx + 0.5*mxNeighborsGPU) % mxNeighborsGPU;
    ry = ry - (int(ry*invlyGPU + 0.5*((ry>0)-(ry<0)))) * lyGPU;
    int jy   = int(ry * invdy + 0.5*myNeighborsGPU) % myNeighborsGPU;
    icel  = jx;
    icel += jy * mxNeighborsGPU;
  }
  np = atomicAdd(&pc->countPartInCellNonBonded[icel],1);
  if(np >= maxNumberPartInCellNonBondedGPU){
    errorKernel[0] = 1;
    errorKernel[4] = 1;
    return;
  }
  pc->partInCellNonBonded[mNeighborsGPU*np+icel] = i;

  // printf("np = %i,  icel = %i \n", np, icel);
}



__global__ void kernelSpreadParticlesForceQuasi2D(const double* rxcellGPU, 
						  const double* rycellGPU, 
						  hipfftDoubleComplex* vxZ,
						  hipfftDoubleComplex* vyZ,
						  const bondedForcesVariables* bFV){
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=(npGPU)) return;   
  
  double fx = 0.0;
  double fy = 0.0;
  double f;

  double rx = fetch_double(texrxboundaryGPU,nboundaryGPU+i);
  double ry = fetch_double(texryboundaryGPU,nboundaryGPU+i);
  
  // INCLUDE EXTERNAL FORCES HERE
  // Example: harmonic potential 
  //  V(r) = (1/2) * k * ((x-x0)**2 + (y-y0)**2 + (z-z0)**2)
  //
  // with spring constant k=0.01
  // and x0=y0=z0=0
  //
  // fx = -0.01*rx;
  // fy = -0.01*ry;
  // fz = -0.01*rz;


  // NEW bonded forces
  if(bondedForcesGPU){
    // call function for bonded forces particle-particle
    forceBondedParticleParticleGPU_2D(i,
				      fx,
				      fy,
				      rx,
				      ry,
				      bFV);
  }
    
  double rxij, ryij, r2;
  int vecino1, vecino2, vecino3, vecino4;
  int vecinopxpy, vecinopxmy;
  int vecinomxpy, vecinomxmy;  
  int icel;
  double r;

  {
    double invdx = double(mxNeighborsGPU)/lxGPU;
    double invdy = double(myNeighborsGPU)/lyGPU;
    r = rx;
    r = r - (int(r*invlxGPU + 0.5*((r>0)-(r<0)))) * lxGPU;
    int jx   = int(r * invdx + 0.5*mxNeighborsGPU) % mxNeighborsGPU;
    r = ry;
    r = r - (int(r*invlyGPU + 0.5*((r>0)-(r<0)))) * lyGPU;
    int jy   = int(r * invdy + 0.5*myNeighborsGPU) % myNeighborsGPU;
    icel  = jx;
    icel += jy * mxNeighborsGPU;
  }
  
  int np;
  if(computeNonBondedForcesGPU){
    //Particles in Cell i
    np = tex1Dfetch(texCountParticlesInCellNonBonded,icel);
    for(int j=0;j<np;j++){
      int particle = tex1Dfetch(texPartInCellNonBonded,mNeighborsGPU*j+icel);
      rxij =  (rx - fetch_double(texrxboundaryGPU,particle));
      rxij =  (rxij - int(rxij*invlxGPU + 0.5*((rxij>0)-(rxij<0)))*lxGPU);
      ryij =  (ry - fetch_double(texryboundaryGPU,particle));
      ryij =  (ryij - int(ryij*invlyGPU + 0.5*((ryij>0)-(ryij<0)))*lyGPU);
      r2 = rxij*rxij + ryij*ryij ;
      f = tex1D(texforceNonBonded1,r2*invcutoff2GPU);
      fx += f * rxij;
      fy += f * ryij;
    }  
    //Particles in Cell vecino1
    vecino1 = tex1Dfetch(texneighbor1GPU, icel);
    np = tex1Dfetch(texCountParticlesInCellNonBonded,vecino1);
    for(int j=0;j<np;j++){
      int particle = tex1Dfetch(texPartInCellNonBonded,mNeighborsGPU*j+vecino1);    
      rxij =  (rx - fetch_double(texrxboundaryGPU,particle));
      rxij =  (rxij - int(rxij*invlxGPU + 0.5*((rxij>0)-(rxij<0)))*lxGPU);
      ryij =  (ry - fetch_double(texryboundaryGPU,particle));
      ryij =  (ryij - int(ryij*invlyGPU + 0.5*((ryij>0)-(ryij<0)))*lyGPU);
      r2 = rxij*rxij + ryij*ryij ;
      f = tex1D(texforceNonBonded1,r2*invcutoff2GPU);
      fx += f * rxij;
      fy += f * ryij;
    }
    //Particles in Cell vecino2
    vecino2 = tex1Dfetch(texneighbor2GPU, icel);
    np = tex1Dfetch(texCountParticlesInCellNonBonded,vecino2);
    for(int j=0;j<np;j++){
      int particle = tex1Dfetch(texPartInCellNonBonded,mNeighborsGPU*j+vecino2);    
      rxij =  (rx - fetch_double(texrxboundaryGPU,particle));
      rxij =  (rxij - int(rxij*invlxGPU + 0.5*((rxij>0)-(rxij<0)))*lxGPU);
      ryij =  (ry - fetch_double(texryboundaryGPU,particle));
      ryij =  (ryij - int(ryij*invlyGPU + 0.5*((ryij>0)-(ryij<0)))*lyGPU);
      r2 = rxij*rxij + ryij*ryij ;
      f = tex1D(texforceNonBonded1,r2*invcutoff2GPU);
      fx += f * rxij;
      fy += f * ryij;
    }
    //Particles in Cell vecino3
    vecino3 = tex1Dfetch(texneighbor3GPU, icel);
    np = tex1Dfetch(texCountParticlesInCellNonBonded,vecino3);
    for(int j=0;j<np;j++){
      int particle = tex1Dfetch(texPartInCellNonBonded,mNeighborsGPU*j+vecino3);    
      rxij =  (rx - fetch_double(texrxboundaryGPU,particle));
      rxij =  (rxij - int(rxij*invlxGPU + 0.5*((rxij>0)-(rxij<0)))*lxGPU);
      ryij =  (ry - fetch_double(texryboundaryGPU,particle));
      ryij =  (ryij - int(ryij*invlyGPU + 0.5*((ryij>0)-(ryij<0)))*lyGPU);
      r2 = rxij*rxij + ryij*ryij ;
      f = tex1D(texforceNonBonded1,r2*invcutoff2GPU);
      fx += f * rxij;
      fy += f * ryij;
    }
    //Particles in Cell vecino4
    vecino4 = tex1Dfetch(texneighbor4GPU, icel);
    //printf("VECINO %i %i \n",icel,vecino4);
    np = tex1Dfetch(texCountParticlesInCellNonBonded,vecino4);
    for(int j=0;j<np;j++){
      int particle = tex1Dfetch(texPartInCellNonBonded,mNeighborsGPU*j+vecino4);    
      rxij =  (rx - fetch_double(texrxboundaryGPU,particle));
      rxij =  (rxij - int(rxij*invlxGPU + 0.5*((rxij>0)-(rxij<0)))*lxGPU);
      ryij =  (ry - fetch_double(texryboundaryGPU,particle));
      ryij =  (ryij - int(ryij*invlyGPU + 0.5*((ryij>0)-(ryij<0)))*lyGPU);
      r2 = rxij*rxij + ryij*ryij ;
      f = tex1D(texforceNonBonded1,r2*invcutoff2GPU);
      fx += f * rxij;
      fy += f * ryij;
    }
    //Particles in Cell vecinopxpy
    vecinopxpy = tex1Dfetch(texneighborpxpyGPU, icel);
    np = tex1Dfetch(texCountParticlesInCellNonBonded,vecinopxpy);
    for(int j=0;j<np;j++){
      int particle = tex1Dfetch(texPartInCellNonBonded,mNeighborsGPU*j+vecinopxpy);    
      rxij =  (rx - fetch_double(texrxboundaryGPU,particle));
      rxij =  (rxij - int(rxij*invlxGPU + 0.5*((rxij>0)-(rxij<0)))*lxGPU);
      ryij =  (ry - fetch_double(texryboundaryGPU,particle));
      ryij =  (ryij - int(ryij*invlyGPU + 0.5*((ryij>0)-(ryij<0)))*lyGPU);
      r2 = rxij*rxij + ryij*ryij ;
      f = tex1D(texforceNonBonded1,r2*invcutoff2GPU);
      fx += f * rxij;
      fy += f * ryij;
    }
    //Particles in Cell vecinopxmy
    vecinopxmy = tex1Dfetch(texneighborpxmyGPU, icel);
    np = tex1Dfetch(texCountParticlesInCellNonBonded,vecinopxmy);
    for(int j=0;j<np;j++){
      int particle = tex1Dfetch(texPartInCellNonBonded,mNeighborsGPU*j+vecinopxmy);
      rxij =  (rx - fetch_double(texrxboundaryGPU,particle));
      rxij =  (rxij - int(rxij*invlxGPU + 0.5*((rxij>0)-(rxij<0)))*lxGPU);
      ryij =  (ry - fetch_double(texryboundaryGPU,particle));
      ryij =  (ryij - int(ryij*invlyGPU + 0.5*((ryij>0)-(ryij<0)))*lyGPU);
      r2 = rxij*rxij + ryij*ryij ;
      f = tex1D(texforceNonBonded1,r2*invcutoff2GPU);
      fx += f * rxij;
      fy += f * ryij;
    }
    //Particles in Cell vecinomxpy
    vecinomxpy = tex1Dfetch(texneighbormxpyGPU, icel);
    np = tex1Dfetch(texCountParticlesInCellNonBonded,vecinomxpy);
    for(int j=0;j<np;j++){
      int particle = tex1Dfetch(texPartInCellNonBonded,mNeighborsGPU*j+vecinomxpy);
      rxij =  (rx - fetch_double(texrxboundaryGPU,particle));
      rxij =  (rxij - int(rxij*invlxGPU + 0.5*((rxij>0)-(rxij<0)))*lxGPU);
      ryij =  (ry - fetch_double(texryboundaryGPU,particle));
      ryij =  (ryij - int(ryij*invlyGPU + 0.5*((ryij>0)-(ryij<0)))*lyGPU);
      r2 = rxij*rxij + ryij*ryij ;
      f = tex1D(texforceNonBonded1,r2*invcutoff2GPU);
      fx += f * rxij;
      fy += f * ryij;
    }
    //Particles in Cell vecinomxmy
    vecinomxmy = tex1Dfetch(texneighbormxmyGPU, icel);
    np = tex1Dfetch(texCountParticlesInCellNonBonded,vecinomxmy);
    for(int j=0;j<np;j++){
      int particle = tex1Dfetch(texPartInCellNonBonded,mNeighborsGPU*j+vecinomxmy);
      rxij =  (rx - fetch_double(texrxboundaryGPU,particle));
      rxij =  (rxij - int(rxij*invlxGPU + 0.5*((rxij>0)-(rxij<0)))*lxGPU);
      ryij =  (ry - fetch_double(texryboundaryGPU,particle));
      ryij =  (ryij - int(ryij*invlyGPU + 0.5*((ryij>0)-(ryij<0)))*lyGPU);
      r2 = rxij*rxij + ryij*ryij ;
      f = tex1D(texforceNonBonded1,r2*invcutoff2GPU);
      fx += f * rxij;
      fy += f * ryij;
    }
  }

  {
    r = rx;
    r = r - (int(r*invlxGPU + 0.5*((r>0)-(r<0)))) * lxGPU;
    int jx   = int(r * invdxGPU + 0.5*mxGPU) % mxGPU;

    r = ry;
    r = r - (int(r*invlyGPU + 0.5*((r>0)-(r<0)))) * lyGPU;
    int jy   = int(r * invdyGPU + 0.5*mytGPU) % mytGPU;

    icel  = jx + jy * mxGPU;
  }

  // printf("rx = %f, ry = %f, icel = %i \n", rx, ry, icel);

  // Loop over neighbor cells
  {
    double rx_distance, ry_distance, norm;
    int kx, ky, kx_neigh, ky_neigh, icel_neigh;
    ky = icel / mxGPU;
    kx = icel % mxGPU;
    // printf("kx = %i,  ky = %i \n", kx, ky);
    for(int ix=-kernelWidthGPU; ix<=kernelWidthGPU; ix++){
      kx_neigh = (kx + ix + mxGPU) % mxGPU;
      rx_distance = rx - (kx_neigh * lxGPU / mxGPU) + lxGPU * 0.5;
      // rx_distance = rx - (kx_neigh * lxGPU / mxGPU) + lxGPU * 0.5 + 0.5 * dxGPU;
      rx_distance = rx_distance - int(rx_distance*invlxGPU + 0.5*((rx_distance>0)-(rx_distance<0)))*lxGPU;

      for(int iy=-kernelWidthGPU; iy<=kernelWidthGPU; iy++){
	ky_neigh = (ky + iy + myGPU) % myGPU;
	icel_neigh = kx_neigh + ky_neigh * mxGPU;

	ry_distance = ry - (ky_neigh * lyGPU / myGPU) + lyGPU * 0.5;
	// ry_distance = ry - (ky_neigh * lyGPU / myGPU) + lyGPU * 0.5 + 0.5 * dyGPU; 
	ry_distance = ry_distance - int(ry_distance*invlyGPU + 0.5*((ry_distance>0)-(ry_distance<0)))*lyGPU;
	r2 = rx_distance*rx_distance + ry_distance*ry_distance;
	norm = GaussianKernel2DGPU(r2, GaussianVarianceGPU);

	/*double dlx, dly;
	{ // For the 3-point kernel
	  dlx = delta(rx_distance);
	  dly = delta(ry_distance);
	  norm = dlx * dly / (volumeGPU);
	  }*/

	atomicAdd(&vxZ[icel_neigh].x, norm * fx);
	atomicAdd(&vyZ[icel_neigh].x, norm * fy);
      }
    }
  } 
}



__global__ void kernelSpreadThermalDriftQuasi2D(const double* rxcellGPU, 
						const double* rycellGPU, 
						hipfftDoubleComplex* vxZ,
						hipfftDoubleComplex* vyZ,
						double *dRand){
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=(npGPU)) return;   
  
  double rx = fetch_double(texrxboundaryGPU,nboundaryGPU+i);
  double ry = fetch_double(texryboundaryGPU,nboundaryGPU+i);
    
  double r2;
  int icel;
  double r;

  {
    r = rx;
    r = r - (int(r*invlxGPU + 0.5*((r>0)-(r<0)))) * lxGPU;
    int jx   = int(r * invdxGPU + 0.5*mxGPU) % mxGPU;

    r = ry;
    r = r - (int(r*invlyGPU + 0.5*((r>0)-(r<0)))) * lyGPU;
    int jy   = int(r * invdyGPU + 0.5*mytGPU) % mytGPU;

    icel   = jx  + jy  * mxGPU;
  }


  // Loop over neighbor cells
  {
    double rx_distance_p, ry_distance_p, rx_distance_m, ry_distance_m;
    double norm;
    int kx, ky, kx_neigh, ky_neigh, icel_neigh;
    ky = icel / mxGPU;
    kx = icel % mxGPU;
    for(int ix=-kernelWidthGPU; ix<=kernelWidthGPU; ix++){
      kx_neigh = (kx + ix + mxGPU) % mxGPU;

      rx_distance_p = (rx + 0.5*deltaRFDGPU*dRand[3*ncellsGPU+i]) - (kx_neigh * lxGPU / mxGPU) + lxGPU * 0.5;
      rx_distance_p = rx_distance_p - int(rx_distance_p*invlxGPU + 0.5*((rx_distance_p>0)-(rx_distance_p<0)))*lxGPU;
      rx_distance_m = (rx - 0.5*deltaRFDGPU*dRand[3*ncellsGPU+i]) - (kx_neigh * lxGPU / mxGPU) + lxGPU * 0.5;
      rx_distance_m = rx_distance_m - int(rx_distance_m*invlxGPU + 0.5*((rx_distance_m>0)-(rx_distance_m<0)))*lxGPU;

      for(int iy=-kernelWidthGPU; iy<=kernelWidthGPU; iy++){
	ky_neigh = (ky + iy + myGPU) % myGPU;
	icel_neigh = kx_neigh + ky_neigh * mxGPU;

	ry_distance_p = (ry + 0.5*deltaRFDGPU*dRand[3*ncellsGPU+npGPU+i]) - (ky_neigh * lyGPU / myGPU) + lyGPU * 0.5;
	ry_distance_p = ry_distance_p - int(ry_distance_p*invlyGPU + 0.5*((ry_distance_p>0)-(ry_distance_p<0)))*lyGPU;
	ry_distance_m = (ry - 0.5*deltaRFDGPU*dRand[3*ncellsGPU+npGPU+i]) - (ky_neigh * lyGPU / myGPU) + lyGPU * 0.5;
	ry_distance_m = ry_distance_m - int(ry_distance_m*invlyGPU + 0.5*((ry_distance_m>0)-(ry_distance_m<0)))*lyGPU;

	// Spread drift kT*S(q+0.5*delta*W)*W
	r2 = rx_distance_p*rx_distance_p + ry_distance_p*ry_distance_p;
	norm = GaussianKernel2DGPU(r2, GaussianVarianceGPU) * temperatureGPU / deltaRFDGPU;

	atomicAdd(&vxZ[icel_neigh].x, norm * dRand[3*ncellsGPU + i]);
	atomicAdd(&vyZ[icel_neigh].x, norm * dRand[3*ncellsGPU + npGPU + i]);

	// Spread drift -kT*S(q-0.5*delta*W)*W
	r2 = rx_distance_m*rx_distance_m + ry_distance_m*ry_distance_m;
	norm = GaussianKernel2DGPU(r2, GaussianVarianceGPU) * temperatureGPU / deltaRFDGPU;

	atomicAdd(&vxZ[icel_neigh].x, -norm * dRand[3*ncellsGPU + i]);
	atomicAdd(&vyZ[icel_neigh].x, -norm * dRand[3*ncellsGPU + npGPU + i]);
      }
    }
  } 
}



__global__ void kernelUpdateVQuasi2D(hipfftDoubleComplex *vxZ, 
				     hipfftDoubleComplex *vyZ){
				     
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=ncellsGPU) return;   

  //Find mode
  int wx, wy;
  wy = i / mxGPU;
  wx = i % mxGPU; 

  if(wx > mxGPU / 2){
    wx -= mxGPU;
  }
  if(wy > myGPU / 2){
    wy -= myGPU;
  }

  double pi = 3.1415926535897932385;
  double kx = wx * 2 * pi / lxGPU;
  double ky = wy * 2 * pi / lyGPU;
  double k_inv = rsqrt(kx*kx + ky*ky);
  double k3_inv = k_inv * k_inv * k_inv;
  hipfftDoubleComplex Wx, Wy;

  if(i == 0 || ((mxGPU % 2) == 0 && (wx == mxGPU / 2)) || ((myGPU % 2) == 0 && (wy == myGPU / 2))){
    vxZ[i].x = 0;
    vxZ[i].y = 0;
    vyZ[i].x = 0;
    vyZ[i].y = 0;
  }
  else{
    Wx.x = (k3_inv / shearviscosityGPU) * (0.5 * ky * (ky*vxZ[i].x - kx*vyZ[i].x) + 0.25 * kx * (kx*vxZ[i].x + ky*vyZ[i].x));
    Wx.y = (k3_inv / shearviscosityGPU) * (0.5 * ky * (ky*vxZ[i].y - kx*vyZ[i].y) + 0.25 * kx * (kx*vxZ[i].y + ky*vyZ[i].y));   
    Wy.x = (k3_inv / shearviscosityGPU) * (0.5 * (-kx) * (ky*vxZ[i].x - kx*vyZ[i].x) + 0.25 * ky * (kx*vxZ[i].x + ky*vyZ[i].x));
    Wy.y = (k3_inv / shearviscosityGPU) * (0.5 * (-kx) * (ky*vxZ[i].y - kx*vyZ[i].y) + 0.25 * ky * (kx*vxZ[i].y + ky*vyZ[i].y));

    vxZ[i].x = Wx.x;
    vxZ[i].y = Wx.y;
    vyZ[i].x = Wy.x;
    vyZ[i].y = Wy.y;
  }
}


__global__ void addStochasticVelocityQuasi2D(hipfftDoubleComplex *vxZ, hipfftDoubleComplex *vyZ, const double *dRand){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=ncellsGPU) return;   

  //Find mode
  int wx, wy, fx, fy, shift;
  int nModes = (ncellsGPU + mxGPU) / 2;
  wy = i / mxGPU;
  wx = i % mxGPU;
  fx = wx;
  fy = wy;
  shift = 0;

  if(wx > mxGPU / 2){
    fx = wx - mxGPU;
    wx = mxGPU - wx;
  }
  if(wy > myGPU / 2){
    fy = wy - myGPU;
    wy = myGPU - wy;
  }
  if(fx * fy < 0){
    shift += nModes / 2;
  }

  int index = wy * mxGPU + wx + shift;
  double pi = 3.1415926535897932385;
  double kx = wx * 2 * pi / lxGPU;
  double ky = wy * 2 * pi / lyGPU;
  double k = sqrt(kx*kx + ky*ky);
  double k3half_inv = rsqrt(k * k * k);
  double sqrtTwo_inv = rsqrt(2.0);
  hipfftDoubleComplex Wx, Wy;
  // double prefactor = sqrt(2.0 * temperatureGPU  / (shearviscosityGPU * dtGPU * lxGPU * lyGPU)) * ncellsGPU;
  double prefactor = fact1GPU;

  // Wx.x = 0.0;
  // Wx.y = 0.0;
  // Wy.x = 0.0;
  // Wy.y = 0.0;

  // Wx.x = prefactor * k3half_inv * (sqrtTwo_inv *   ky  * dRand[index]            + 0.5 * kx * dRand[nModes   + index]);
  // Wy.x = prefactor * k3half_inv * (sqrtTwo_inv * (-kx) * dRand[nModes*2 + index] + 0.5 * ky * dRand[nModes*3 + index]);
  // Wx.y = prefactor * k3half_inv * (sqrtTwo_inv *   ky  * dRand[nModes*4 + index] + 0.5 * kx * dRand[nModes*5 + index]);
  // Wy.y = prefactor * k3half_inv * (sqrtTwo_inv * (-kx) * dRand[nModes*6 + index] + 0.5 * ky * dRand[nModes*7 + index]); 

  Wx.x = prefactor * k3half_inv * (1.0 * sqrtTwo_inv *   ky  * dRand[           index] + 0.5 * kx * dRand[nModes   + index]);
  Wy.x = prefactor * k3half_inv * (1.0 * sqrtTwo_inv * (-kx) * dRand[nModes*2 + index] + 0.5 * ky * dRand[nModes*3 + index]);
  Wx.y = prefactor * k3half_inv * (1.0 * sqrtTwo_inv *   ky  * dRand[nModes*4 + index] + 0.5 * kx * dRand[nModes*5 + index]);
  Wy.y = prefactor * k3half_inv * (1.0 * sqrtTwo_inv * (-kx) * dRand[nModes*6 + index] + 0.5 * ky * dRand[nModes*7 + index]); 

  if(fx < 0){
    Wx.y *= -1.0;
    Wy.y *= -1.0;
  }

  /*if(wx == 0 and wy == 2){
    printf("Initial values wx = %i, wy = %i, Used values wx = %i, wy = %i, shift = %i, index = %i, fx = %i, Wx.y = %f \n", i % mxGPU, i / mxGPU, wx, wy, shift, index, fx, Wx.y);
    }*/
 
  if(((mxGPU % 2) == 0 && (wx == mxGPU / 2)) || ((myGPU % 2) == 0 && (wy == myGPU / 2)) || (i == 0)){
    Wx.x = 0;
    Wx.y = 0;
    Wy.x = 0;
    Wy.y = 0;
  }
  else if(wx == 0 or wy == 0){
    Wx.y = 0;
    Wy.y = 0;
  }
  else{
    vxZ[i].x += Wx.x;
    vxZ[i].y += Wx.y;
    vyZ[i].x += Wy.x;
    vyZ[i].y += Wy.y;
  }
}

__global__ void updateParticlesQuasi2D(particlesincell* pc, 
				       int* errorKernel,
				       const double* rxcellGPU,
				       const double* rycellGPU,
				       double* rxboundaryGPU,  // q^{} to interpolate
				       double* ryboundaryGPU, 
				       double* rxboundaryPredictionGPU,  // q^{udpdated}
				       double* ryboundaryPredictionGPU, 
				       const double* vxGPU,
				       const double* vyGPU,
				       double dt){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=(npGPU)) return;   
  
  // double rx = fetch_double(texrxboundaryGPU,nboundaryGPU+i);
  // double ry = fetch_double(texryboundaryGPU,nboundaryGPU+i);
  double rx = rxboundaryGPU[i];
  double ry = ryboundaryGPU[i];

  // printf("rx = %f, ry = %f \n", rx, ry);
					 
  double r;
  int icel;

  // Particle location in grid cells
  {
    r = rx;
    r = r - (int(r*invlxGPU + 0.5*((r>0)-(r<0)))) * lxGPU;
    int jx   = int(r * invdxGPU + 0.5*mxGPU) % mxGPU;

    r = ry;
    r = r - (int(r*invlyGPU + 0.5*((r>0)-(r<0)))) * lyGPU;
    int jy   = int(r * invdyGPU + 0.5*mytGPU) % mytGPU;

    icel  = jx + jy * mxGPU;
  }

  // Interpolate fluid velocity
  double ux = 0.0, uy = 0.0;

  // Loop over neighbor cells
  {
    double rx_distance, ry_distance, norm, r2;
    int kx, ky, kx_neigh, ky_neigh, icel_neigh;
    ky = icel / mxGPU;
    kx = icel % mxGPU;
    // double icel_double = double(icel);
    for(int ix=-kernelWidthGPU; ix<=kernelWidthGPU; ix++){
      kx_neigh = (kx + ix + mxGPU) % mxGPU;
      rx_distance = rx - (kx_neigh * lxGPU / mxGPU) + lxGPU * 0.5;
      // rx_distance = rx - (kx_neigh * lxGPU / mxGPU) + lxGPU * 0.5 + 0.5 * dxGPU;
      rx_distance = rx_distance - int(rx_distance*invlxGPU + 0.5*((rx_distance>0)-(rx_distance<0)))*lxGPU;

      for(int iy=-kernelWidthGPU; iy<=kernelWidthGPU; iy++){
	ky_neigh = (ky + iy + myGPU) % myGPU;
	icel_neigh = kx_neigh + ky_neigh * mxGPU;

	ry_distance = ry - (ky_neigh * lyGPU / myGPU) + lyGPU * 0.5;
	// ry_distance = ry - (ky_neigh * lyGPU / myGPU) + lyGPU * 0.5 + 0.5 * dyGPU;
	ry_distance = ry_distance - int(ry_distance*invlyGPU + 0.5*((ry_distance>0)-(ry_distance<0)))*lyGPU;
	r2 = rx_distance*rx_distance + ry_distance*ry_distance;
	norm = GaussianKernel2DGPU(r2, GaussianVarianceGPU);

	/*double dlx, dly;
	{ // For the 3-point kernel
	  dlx = delta(rx_distance);
	  dly = delta(ry_distance);
	  norm = dlx * dly;
	  }*/

	ux += norm * vxGPU[icel_neigh];
	uy += norm * vyGPU[icel_neigh];
      }
    }
  }

  double volumeCell = dxGPU * dyGPU;
  // printf("i = %i, ux = %e, uy = %e, dt = %e \n", i, volumeCell * ux, volumeCell * uy, dt);

  // rxboundaryGPU[i] += volumeCell * ux * dt;
  // ryboundaryGPU[i] += volumeCell * uy * dt;
  rxboundaryPredictionGPU[i] = fetch_double(texrxboundaryGPU,nboundaryGPU+i) + volumeCell * ux * dt;
  ryboundaryPredictionGPU[i] = fetch_double(texryboundaryGPU,nboundaryGPU+i) + volumeCell * uy * dt;

}



__global__ void kernelUpdateVIncompressibleSpectral2D(hipfftDoubleComplex *vxZ, 
						      hipfftDoubleComplex *vyZ,
						      hipfftDoubleComplex *vzZ, 
						      hipfftDoubleComplex *WxZ, 
						      hipfftDoubleComplex *WyZ, 
						      hipfftDoubleComplex *WzZ, 
						      prefactorsFourier *pF){
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=ncellsGPU) return;   

  // Find mode
  int wx, wy;
  wy = i / mxGPU;
  wx = i % mxGPU;

  if(wx > mxGPU / 2){
    wx -= mxGPU;
  }
  if(wy > myGPU / 2){
    wy -= myGPU;
  }

  double pi = 3.1415926535897932385;
  double kx = wx * 2.0 * pi / lxGPU;
  double ky = wy * 2.0 * pi / lyGPU;
  
  // Construct L
  double L;
  L = -(kx * kx) - (ky * ky);
  
  // Construct denominator
  double denominator = -shearviscosityGPU * L;
  
  // Construct GW
  hipfftDoubleComplex GW;
  GW.x = kx * WxZ[i].x + ky * WyZ[i].x ;
  GW.y = kx * WxZ[i].y + ky * WyZ[i].y ;
  
  if((i == 0) || ((mxGPU % 2) == 0 && (wx == mxGPU / 2)) || ((myGPU % 2) == 0 && (wy == myGPU / 2))){
    vxZ[i].x = 0; 
    vxZ[i].y = 0; 
    vyZ[i].x = 0; 
    vyZ[i].y = 0; 
  }
  else{
    vxZ[i].x = (WxZ[i].x + kx * GW.x / L) / denominator;
    vxZ[i].y = (WxZ[i].y + kx * GW.y / L) / denominator;
    vyZ[i].x = (WyZ[i].x + ky * GW.x / L) / denominator;
    vyZ[i].y = (WyZ[i].y + ky * GW.y / L) / denominator;
  }
}




__global__ void kernelUpdateVIncompressibleStokes2D(hipfftDoubleComplex *vxZ, 
						    hipfftDoubleComplex *vyZ,
						    hipfftDoubleComplex *vzZ, 
						    hipfftDoubleComplex *WxZ, 
						    hipfftDoubleComplex *WyZ, 
						    hipfftDoubleComplex *WzZ, 
						    prefactorsFourier *pF){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=ncellsGPU) return;   

  //Find mode
  int wx, wy;
  wy = i / mxGPU;
  wx = i % mxGPU;

  double kx, ky;
  kx = pF->gradKx[wx].y;
  ky = pF->gradKy[wy].y;

  /*if(wx > mxGPU / 2){
    kx = -1.0 * kx;
  }
  if(wy > myGPU / 2){
    ky = -1.0 * ky;
    }*/
  
  //Construct L
  double L;
  L = -kx*kx - ky*ky;

  //Construct denominator
  double denominator = -shearviscosityGPU * L;

  //Construct GW
  hipfftDoubleComplex GW;
  GW.x = kx * WxZ[i].x + ky * WyZ[i].x ;
  GW.y = kx * WxZ[i].y + ky * WyZ[i].y ;
  
  if(i==0){
    vxZ[i].x = 0; // WxZ[i].x;
    vxZ[i].y = 0; // WxZ[i].y;
    vyZ[i].x = 0; // WyZ[i].x;
    vyZ[i].y = 0; // WyZ[i].y;
  }
  else{
    vxZ[i].x = (WxZ[i].x + kx * GW.x / L) / denominator;
    vxZ[i].y = (WxZ[i].y + kx * GW.y / L) / denominator;
    vyZ[i].x = (WyZ[i].x + ky * GW.x / L) / denominator;
    vyZ[i].y = (WyZ[i].y + ky * GW.y / L) / denominator;
  }
  
}
